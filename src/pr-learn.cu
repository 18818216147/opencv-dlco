#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2014  Balint Cristian (cristian dot balint at gmail dot com)
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * * Redistributions of source code must retain the above copyright notice, this
 *   list of conditions and the following disclaimer.
 *
 * * Redistributions in binary form must reproduce the above copyright notice,
 *   this list of conditions and the following disclaimer in the documentation
 *   and/or other materials provided with the distribution.
 *
 * * Neither the name of copyright holder nor the names of its
 *   contributors may be used to endorse or promote products derived from
 *   this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/* pr-learn.cu */
/* PR lean cuda kernels */

#include "stdio.h"

#include <opencv2/core/core.hpp>
#include "opencv2/cudev.hpp"
#include "opencv2/core/cuda/common.hpp"
#include "opencv2/core/cuda/utility.hpp"


using namespace cv::cuda;
using namespace cv::cudev;

namespace cv { namespace cuda {
namespace dlco {

__global__ static void kSubtractVectorsByRows( const GlobPtrSz<float> src1,
                                               const GlobPtrSz<float> src2,
                                               GlobPtr<float> dst )
{
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    while ( idx < src1.rows )
    {
      register float tsum = 0;
      for ( int i = 0; i < src2.rows; i++ )
      {
        register float rsum = src1.data[ idx ] + 1 - src2.data[ i ];
        tsum += (rsum > 0) ? rsum : 0;
      }
      dst.data[idx] = tsum;
      idx += gridDim.x * blockDim.x;
    }
}

void SubtractVectorsByRows( const cuda::GpuMat& src1, const cuda::GpuMat& src2, cuda::GpuMat& dst, Stream& _stream )
{
    const dim3 grid ( 4096, 1, 1 );
    const dim3 block(  512, 1, 1 );

    hipStream_t stream = StreamAccessor::getStream(_stream);

    dst = cuda::GpuMat( src1.rows, 1, CV_32F, Scalar::all(0) );
    kSubtractVectorsByRows<<< grid, block, 0, stream >>>( globPtr<float>(src1), globPtr<float>(src2), globPtr<float>(dst) );

    CV_CUDEV_SAFE_CALL( hipGetLastError() );
    CV_CUDEV_SAFE_CALL( hipDeviceSynchronize() );
}

} // end namespace dlco
}} // end namespaces
